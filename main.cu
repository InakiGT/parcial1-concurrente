
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void calcDeterminante(float * matriz, float * determinante, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        determinante[0] += matriz[idx] * (
            matriz[((0 + 1) % N) * N + ((idx + 1) % N)] * matriz[((0 + 2) % N) * N + ((idx + 2) % N)] -
            matriz[((0 + 1) % N) * N + ((idx + 2) % N)] * matriz[((0 + 2) % N) * N + ((idx + 1) % N)]
        );
    }
}

__global__ void matrizAdjunta(float * matriz, float * adjunta, int N) {
    int fila    = blockIdx.y * blockDim.y + threadIdx.y;
    int columna = blockIdx.x * blockDim.x + threadIdx.x;

    if (fila < N && columna < N) {
        int indice = columna * N + fila;
        adjunta[indice] = (
            matriz[((fila + 1) % N) * N + ((columna + 1) % N)] * matriz[((fila + 2) % N) * N + ((columna + 2) % N)] -
            matriz[((fila + 1) % N) * N + ((columna + 2) % N)] * matriz[((fila + 2) % N) * N + ((columna + 1) % N)]
        );

        if ((fila + columna) % 2 == 1) {
            adjunta[indice] = -adjunta[indice];
        }
    }
}

__global__ void calcInversa(float * adjunta, float * inversa, float * determinante, int N) {
    int fila    = blockIdx.y * blockDim.y + threadIdx.y;
    int columna = blockIdx.x * blockDim.x + threadIdx.x;

    if (fila < N && columna < N) {
        int indice = fila * N + columna;

        inversa[indice] = adjunta[indice] / determinante[0];
    }
}

int main(int argc, char * argv[]) {

    int N = atoi(argv[1]);
    srand(time(NULL));

    float * matriz          = (float *) malloc(N * N * sizeof(float));
    float * inversa         = (float *) malloc(N * N * sizeof(float));
    float * determinante_h  = (float *) malloc(N * sizeof(float));

    for (int i = 0; i < N * N; i++)
        matriz[i] = rand() % 100;

    printf("Matriz original: \n");
    for (int i = 0; i < N * N; i++)
        printf("%f ", matriz[i]);

    float *matriz_d, *determinante_d, *adjunta_d, *inversa_d;

    hipMalloc(&matriz_d, N * N * sizeof(float));
    hipMalloc(&adjunta_d, N * N * sizeof(float));
    hipMalloc(&inversa_d, N * N * sizeof(float));
    hipMalloc(&determinante_d, N * sizeof(float));

    hipMemcpy(matriz_d, matriz, N * N * sizeof(float), hipMemcpyHostToDevice);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    int tambloque   = prop.maxThreadsPerBlock;
    int numbloques  = (N + tambloque - 1) / tambloque;

    calcDeterminante<<<numbloques, tambloque>>>(matriz_d, determinante_d, N);

    hipMemcpy(determinante_h, determinante_d, N * sizeof(float), hipMemcpyDeviceToHost);

    printf("PEPE MADERO: %f", determinante_h[0]);
    if (determinante_h[0] == 0) {
        printf("\nLa matiz no es invertible");
    } else {
        numbloques = (N * N + tambloque - 1) / tambloque;

        dim3 tamBloque(numbloques, numbloques); 
        dim3 tamMalla((N * N + numbloques - 1) / numbloques, (N * N + numbloques - 1) / numbloques);

        matrizAdjunta<<<tamMalla, tamBloque>>>(matriz_d, adjunta_d, N);
        calcInversa<<<tamMalla, tamBloque>>>(adjunta_d, inversa_d, determinante_d, N);

        hipMemcpy(inversa, inversa_d, N * N * sizeof(float), hipMemcpyDeviceToHost);

        printf("Matriz inversa: \n");
        for (int i = 0; i < N * N; i++)
            printf("%f ", inversa[i]);
    }

    hipDeviceSynchronize();

    free(matriz);
    free(inversa);
    free(determinante_h);
    hipFree(matriz_d);
    hipFree(adjunta_d);
    hipFree(inversa_d);
    hipFree(determinante_d);

    return 0;
}